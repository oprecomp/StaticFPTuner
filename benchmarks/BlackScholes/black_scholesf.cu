
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#define A1 0.31938153f
#define A2 -0.356563782f
#define A3 1.781477937f
#define A4 -1.821255978f
#define A5 1.330274429f
#define RSQRT2PI 0.3989422804f

__device__ float cndGPU(float d)
{
    float
        K = 1.0f / (1.0f + 0.2316419f * fabsf(d));

    float
        cnd = RSQRT2PI * expf(- 0.5f * d * d) * 
        (K * (A1 + K * (A2 + K * (A3 + K * (A4 + K * A5)))));

    if(d > 0)
        cnd = 1.0f - cnd;

    return cnd;
}


///////////////////////////////////////////////////////////////////////////////
// Black-Scholes formula for both call and put
///////////////////////////////////////////////////////////////////////////////
__device__ void BlackScholesBodyGPU
(
    float& CallResult,
    float& PutResult,
    float S, //Stock price
    float X, //Option strike
    float T, //Option years
    float R, //Riskless rate
    float V  //Volatility rate
)
{
    float sqrtT, expRT;
    float d1, d2, CNDD1, CNDD2;

    sqrtT = sqrtf(T);
    d1 = (logf(S / X) + (R + 0.5f * V * V) * T) / (V * sqrtT);
    d2 = d1 - V * sqrtT;


    CNDD1 = cndGPU(d1);
    CNDD2 = cndGPU(d2);
    //printf("%.15f,", CNDD1);

    //Calculate Call and Put simultaneously
    expRT = expf(- R * T);
    CallResult = S * CNDD1 - X * expRT * CNDD2;
    PutResult  = X * expRT * (1.0f - CNDD2) - S * (1.0f - CNDD1);
}


////////////////////////////////////////////////////////////////////////////////
//Process an array of optN options on GPU
////////////////////////////////////////////////////////////////////////////////
__global__ void BlackScholesGPU(
    float *d_CallResult,
    float *d_PutResult,
    float *d_StockPrice,
    float *d_OptionStrike,
    float *d_OptionYears,
    float Riskfree,
    float Volatility,
    int optN
)
{
    //Thread index
    const int      tid = blockDim.x * blockIdx.x + threadIdx.x;
    //Total number of threads in execution grid
    const int THREAD_N = blockDim.x * gridDim.x;

    //No matter how small is execution grid or how large OptN is,
    //exactly OptN indices will be processed with perfect memory coalescing
    for(int opt = tid; opt < optN; opt += THREAD_N)
        BlackScholesBodyGPU(
            d_CallResult[opt],
            d_PutResult[opt],
            d_StockPrice[opt],
            d_OptionStrike[opt],
            d_OptionYears[opt],
            Riskfree,
            Volatility
        );
}


float RandFloat(float low, float high){
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}

#define OPT_N  400000


const int  NUM_ITERATIONS = 512;


const int          OPT_SZ = OPT_N * sizeof(float);
const float      RISKFREE = 0.02;
const float    VOLATILITY = 0.30;


int main()
{
    float * h_CallResultGPU = (float *)malloc(OPT_SZ);
    float * h_PutResultGPU  = (float *)malloc(OPT_SZ);
    float * h_StockPrice    = (float *)malloc(OPT_SZ);
    float * h_OptionStrike  = (float *)malloc(OPT_SZ);
    float * h_OptionYears   = (float *)malloc(OPT_SZ);


    float
        //Results calculated by GPU
        *d_CallResult,
        *d_PutResult,
        //GPU instance of input data
        *d_StockPrice,
        *d_OptionStrike,
        *d_OptionYears;

    hipMalloc((void **)&d_CallResult,   OPT_SZ);
    hipMalloc((void **)&d_PutResult,    OPT_SZ);
    hipMalloc((void **)&d_StockPrice,   OPT_SZ);
    hipMalloc((void **)&d_OptionStrike, OPT_SZ);
    hipMalloc((void **)&d_OptionYears,  OPT_SZ);   

    srand(5347);

    //Generate options set
    int i;
    for(i = 0; i < OPT_N; i++)
    {
        h_CallResultGPU[i] = 0.0;
        h_PutResultGPU[i]  = -1.0;
        h_StockPrice[i]    = RandFloat(5.0f, 30.0f);
        h_OptionStrike[i]  = RandFloat(1.0f, 100.0f);
        h_OptionYears[i]   = RandFloat(0.25f, 10.0f);
    }

    hipMemcpy(d_StockPrice,  h_StockPrice,   OPT_SZ, hipMemcpyHostToDevice);
    hipMemcpy(d_OptionStrike, h_OptionStrike,  OPT_SZ, hipMemcpyHostToDevice);
    hipMemcpy(d_OptionYears,  h_OptionYears,   OPT_SZ, hipMemcpyHostToDevice);  

    for(i = 0; i < NUM_ITERATIONS; i++){
        BlackScholesGPU<<<256, 128>>>(
            d_CallResult,
            d_PutResult,
            d_OptionStrike,
            d_StockPrice,
            d_OptionYears,
            RISKFREE,
            VOLATILITY,
            OPT_N
        );      
    }

    hipMemcpy(h_CallResultGPU, d_CallResult, OPT_SZ, hipMemcpyDeviceToHost);
    hipMemcpy(h_PutResultGPU,  d_PutResult,  OPT_SZ, hipMemcpyDeviceToHost);   

    //for(i = 0; i < OPT_N; i++)
    //    printf("%.15f,", h_CallResultGPU[i]);

  return 0;
}